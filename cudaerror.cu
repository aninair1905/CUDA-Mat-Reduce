#include <stdlib.h>
#include <errno.h>
#include "error.h"
#include "cudaerror.h"

void ecudaMalloc(void** devptr, size_t size) {
    errno = 0;
    hipError_t res = hipMalloc(devptr, size);

    if (res != hipSuccess) {
        print_error_msg("call to hipMalloc failed: %s", hipGetErrorString(res));
        exit(-1);
    }

    return ;
}

void ecudaMemcpy(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) {
    errno = 0;
    hipError_t res = hipMemcpy(dst, src, count, kind);

    if (res != hipSuccess) {
        print_error_msg("call to hipMemcpy failed: %s", hipGetErrorString(res));
        exit(-1);
    }

    return ;
}

void ecudaFree(void* devptr) {
    errno = 0;
    hipError_t res = hipFree(devptr);

    if (res != hipSuccess) {
        print_error_msg("call to hipMemcpy failed: %s", hipGetErrorString(res));
        exit(-1);
    }

    return ;
}
